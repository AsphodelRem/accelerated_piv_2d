#include <math/fft_handlers.cuh>
#include <math/additional.cuh>

FFTHandler::FFTHandler(const PIVParameters &parameters) :parameters_(parameters)
{
  const int segment_size = parameters.image_parameters.window_size;

  rank = 2;
  n[0] = n[1] = segment_size;
  i_dist = segment_size * segment_size;
  o_dist = segment_size * (segment_size / 2 + 1);
  in_embed[0] = in_embed[1] = segment_size;
  on_embed[0] = segment_size;
  on_embed[1] = segment_size / 2 + 1;
  stride = 1;
  batch_size = parameters.image_parameters.GetNumberOfWindows();
}

ForwardFFTHandler::ForwardFFTHandler(const PIVParameters &parameters) : FFTHandler(parameters)
{
  hipfftPlanMany(&cufft_handler_, rank, n, in_embed, stride,
    i_dist, on_embed, stride, o_dist, HIPFFT_R2C, batch_size);

  auto [height, width] = this->parameters_.image_parameters.GetSpectrumSize();
  const int spectrum_height = height;
  const int spectrum_width = width;

  this->result = make_shared_gpu<hipComplex>(spectrum_height * spectrum_width);
}

void ForwardFFTHandler::ComputeForwardFFT(const SharedPtrGPU<float> &image, bool to_conjugate)
{
  hipfftExecR2C(this->cufft_handler_, image.get(), this->result.get());

  if (to_conjugate)
  {
    Conjugate(this->result.get(), this->parameters_.image_parameters.height,
              this->parameters_.image_parameters.width);
  }
}

ForwardFFTHandler &
ForwardFFTHandler::operator*=(const ForwardFFTHandler &other)
{
  auto [height, width] = this->parameters_.image_parameters.GetSpectrumSize();
  const int spectrum_height = height;
  const int spectrum_width = width;

  ElementwiseMultiplication(this->result.get(), other.result.get(), spectrum_height, spectrum_width);

  return *this;
}

BackwardFFTHandler::BackwardFFTHandler(const PIVParameters &parameters) : FFTHandler(parameters)
{
  hipfftPlanMany(&cufft_handler_, rank, n, on_embed, stride, o_dist, in_embed, stride, i_dist, HIPFFT_C2R, batch_size);
  this->result = make_shared_gpu<float>(parameters.image_parameters.height * parameters.image_parameters.width);
  this->buffer_ = make_shared_gpu<float>(parameters.image_parameters.height * parameters.image_parameters.width);
}

void BackwardFFTHandler::ComputeBackwardFFT(const SharedPtrGPU<hipComplex> &image)
{
  hipfftExecC2R(this->cufft_handler_, image.get(), this->buffer_.get());

  ShiftSpectrum(this->buffer_.get(), this->result.get(),
           this->parameters_.image_parameters.window_size,
           this->parameters_.image_parameters.window_size,
           this->parameters_.image_parameters.GetNumberOfWindows());
}

