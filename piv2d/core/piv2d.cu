#include "hip/hip_runtime.h"
#include <piv2d.cuh>

PIVDataContainer::PIVDataContainer(PIVParameters &parameters) : parameters_(parameters)
{
    auto number_of_window = parameters.image_parameters.GetNumberOfWindows();

    this->data = std::make_shared<Point2D<float>[]>(number_of_window);
    this->preprocessed_data_ = make_shared_gpu<Point2D<float>>(number_of_window);
}

__global__
void FindMovements_kernel(Point2D<float> *interpolated_coordinates,
                                    Point2D<float> *output_speed,
                                    unsigned int length,
                                    float scale_factor,
                                    float time,
                                    unsigned int window_size,
                                    bool to_physical_view = false)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length)
    {
        bool is_there_movements_by_x = (interpolated_coordinates[idx].x != 0);
        bool is_there_movements_by_y = (interpolated_coordinates[idx].y != 0);

        output_speed[idx].x = is_there_movements_by_x * (interpolated_coordinates[idx].x - (window_size / 2)) * scale_factor / time;
        output_speed[idx].y = is_there_movements_by_y * (interpolated_coordinates[idx].y - (window_size / 2)) * scale_factor / time;

        if (to_physical_view)
        {
            output_speed[idx].y = -output_speed[idx].y;
        }
    }
}

void FindMovements(SharedPtrGPU<Point2D<float>> &input, SharedPtrGPU<Point2D<float>> &output, PIVParameters &parameters)
{
    auto length = parameters.image_parameters.GetNumberOfWindows();

    dim3 grid_size = {(length + 127) / 128};
    dim3 threads_per_block = {128};

    FindMovements_kernel<<<grid_size, threads_per_block>>>(input.get(),
                                                        output.get(),
                                                        length,
                                                        1, 1,
                                                        parameters.image_parameters.window_size);
}

void PIVDataContainer::StoreData(SharedPtrGPU<Point2D<float>> &data)
{
    FindMovements(data, preprocessed_data_, parameters_);

    preprocessed_data_.CopyDataToHost(this->data.get());
}

PIVDataContainer StartPIV2D(ImageContainer &container, PIVParameters &parameters)
{
    ForwardFFTHandler fourier_image_1(parameters);
    ForwardFFTHandler fourier_image_2(parameters);

    BackwardFFTHandler correlation_function(parameters);

    Filter filter(parameters);

    MultiArgMaxSearch multi_max_search(parameters);

    Interpolation interpolation(parameters);

    PIVDataContainer data(parameters);

    while (!container.IsEmpty())
    {
        auto new_data = container.GetImages();

        fourier_image_1.ComputeForwardFFT(new_data.GetFirstImage(), true);
        fourier_image_2.ComputeForwardFFT(new_data.GetSecondImage());

        fourier_image_1 *= fourier_image_2;

        filter.filter(fourier_image_1.result);

        correlation_function.ComputeBackwardFFT(fourier_image_1.result);

        multi_max_search.GetMaxForAllWindows(correlation_function.result);

        interpolation.Interpolate(correlation_function.result, multi_max_search.result);

        data.StoreData(interpolation.result);
    }

    return data;
}
